#include "hip/hip_runtime.h"
#include "./resize_layer.hpp"  
#include "../util/math_functions.hpp"  
#include <vector>
#include <opencv2/opencv.hpp>  
#include<iostream>
using namespace std;
namespace caffe {  


__global__ void kernel_ResizeBlob(const int nthreads,const int num,const int channels, const float* src, const int src_height, const int src_width,
		float* dst, const int dst_height, const int dst_width, const float scale_h, const float scale_w)
{
	CUDA_KERNEL_LOOP(index, nthreads) {
		int i = index %( dst_height * dst_width);
		int c = (index/(dst_height * dst_width))%channels;
		int n = (index/(dst_height * dst_width))/channels;
		int src_offset = (n * channels + c) * src_height * src_width;
		int dst_offset = (n * channels + c) * dst_height * dst_width;

		const float* src_data = src+src_offset;
		float* dst_data = dst+dst_offset;

		int dst_h = i /dst_width;
		float fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const float w_h0 = std::abs(1.0f - fh);
		const float w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = i %dst_width;
		float fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const float w_w0 = std::abs(1.0f - fw);
		const float w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;


		const int src_idx = src_offset_1 + src_w;
		float res = (w_h0 * w_w0 * src_data[src_idx]);

		if (src_w + 1 < src_width)
			res += (w_h0 * w_w1 * src_data[src_idx + 1]);
		if (src_h + 1 < src_height)
			res += (w_h1 * w_w0 * src_data[src_idx + src_width]);

		if (src_w + 1 < src_width && src_h + 1 < src_height)
			res += (w_h1 * w_w1 * src_data[src_idx + src_width + 1]);

		dst_data[dst_idx] = res;
	}
}



void ResizeLayer::Forward_gpu(  
    const vector<Blob*>& bottom, const vector<Blob*>& top) {   
	const int src_num = bottom[0]->num();
	const int src_channels = bottom[0]->channels();
	const int src_height = bottom[0]->height();
	const int src_width = bottom[0]->width();

	const int dst_channels = top[0]->channels();
	const int dst_height = top[0]->height();
	const int dst_width = top[0]->width();
	
	const float scale_w = src_width / (float)dst_width;
	const float scale_h = src_height / (float)dst_height;
	int loop_n = dst_height * dst_width*dst_channels*src_num;
	const float* src_data = bottom[0]->gpu_data();
	float* dst_data = top[0]->mutable_gpu_data();
	kernel_ResizeBlob<<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(loop_n,src_num,src_channels,src_data, src_height,src_width,dst_data, dst_height, dst_width,scale_h,scale_w);  
	CUDA_POST_KERNEL_CHECK;
}  

}  // namespace caffe  